
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>

// Addition of arrays using a stride loop

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

__global__
void add(int n, double* x, double const* y)
{
   int index = threadIdx.x;
   int stride = blockDim.x;
   for (int i = index; i < n; i += stride)
   {
      x[i] = x[i] + y[i];
   }
}

int main()
{
   int N = 1<<20; // pow(2,20) = 1,048,576

   // allocate memory
   double* x;
   checkError(hipMallocManaged(&x, N*sizeof(double)));

   double* y;
   checkError(hipMallocManaged(&y, N*sizeof(double)));

   // initialize arrays
   for (int i = 0; i < N; i++)
   {
      x[i] = 1.0;
      y[i] = 2.0;
   }

   auto t1 = std::chrono::high_resolution_clock::now();

   add<<<1, 1>>>(N, x, y);
   checkError(hipDeviceSynchronize());

   auto t2 = std::chrono::high_resolution_clock::now();

   auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2-t1).count();

   std::cout << "Time = " << duration << " us\n";

   // clean up
   hipFree(x);
   hipFree(y);
}


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>

// Addition of arrays using a stride loop

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

__global__
void add(int n, double* x, double const* y)
{
   int index = blockIdx.x*blockDim.x + threadIdx.x;
   int stride = blockDim.x*gridDim.x;
   for (int i = index; i < n; i += stride)
   {
      x[i] = x[i] + y[i];
   }
}

int main()
{
   int N = 1<<20; // pow(2,20) = 1,048,576

   // allocate memory on the host
   double* x = new double[N];
   double* y = new double[N];

   // initialize arrays
   for (int i = 0; i < N; i++)
   {
      x[i] = 1.0;
      y[i] = 2.0;
   }

   // allocate memory on the device
   double* xDevice;
   double* yDevice;
   checkError(hipMalloc(&xDevice, N*sizeof(double)));
   checkError(hipMalloc(&yDevice, N*sizeof(double)));

   // copy memory from host to device
   checkError(hipMemcpy(xDevice, x, N*sizeof(double), hipMemcpyHostToDevice));
   checkError(hipMemcpy(yDevice, y, N*sizeof(double), hipMemcpyHostToDevice));

   int Threads = 256;
   int Blocks = (N+Threads-1)/Threads;

   auto t1 = std::chrono::high_resolution_clock::now();

   add<<<Blocks, Threads>>>(N, xDevice, yDevice);
   checkError(hipDeviceSynchronize());

   auto t2 = std::chrono::high_resolution_clock::now();

   // copy memory from device back to host
   checkError(hipMemcpy(x, xDevice, N*sizeof(double), hipMemcpyDeviceToHost));

   auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2-t1).count();

   std::cout << "Time = " << duration << " us\n";

   // clean up
   hipFree(x);
   hipFree(y);
}
